/*
        Grupo: 8
        Nome: Abner Eduardo Silveira Santos             NUSP: 10692012
        Nome: Gyovana Mayara Moriyama                   NUSP: 10734387
        Nome: Henrique Matarazo Camillo                 NUSP: 10294943
        Nome: Vitor Augusto de Oliveira                 NUSP: 9360815
*/

/*
Soma dois vetores
Ilustra a alocação dinâmica da memoria compartilhada
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <limits.h>

#define TAM 16
#define TPB 2

__global__ void min_max(int *vetorA, int *vetorB,int *min_max_val,int tam)
{ 
    int idx = blockDim.x * blockIdx.x + threadIdx.x; 
    if (idx < tam) 
    {
        if (vetorA[idx] < min_max_val[0])
            min_max_val[0] = vetorA[idx];
        if (vetorA[idx] > min_max_val[1])
            min_max_val[1] = vetorA[idx];
        if (vetorB[idx] < min_max_val[0])
            min_max_val[0] = vetorB[idx];
        if (vetorB[idx] > min_max_val[1])
            min_max_val[1] = vetorB[idx];
    }
}

__global__ void mult(int *vetorA, int *vetorB,int *vetorC,int tam)
{ 
    int idx = blockDim.x * blockIdx.x + threadIdx.x; 
    if (idx < tam) 
    {
        vetorC[idx] = vetorA[idx] * vetorB[idx];
    }
}

// __global__ void soma(int *vetorA, int *vetorB,int *vetorC,int tam)
// { 
//     int idx = blockDim.x * blockIdx.x + threadIdx.x; 
//     if (idx < tam) 
//     {
//         vetorC[idx]=vetorA[idx]+vetorB[idx];
//     }
// }

int main(int argc,char **argv){

    //Declara as matrizes
    int *A,*B,*C;
    //Declara as variáveis de índice
    int i,j,k,dim;
    //Declara o acumulador para o produto escalar global
    int somape, minimo = INT_MAX, maximo = INT_MIN;
    int* min_max_val;

    //Declara um vetor para os produtos escalares locais
    // int prod_escalar;

    FILE *inputfile;     // handler para o arquivo de entrada
    char *inputfilename;  // nome do arquivo de entrada

   
    if (argc < 2)
    {
        printf("Please run with input file name, i.e., num_perf_mpi inputfile.ext\n");
        exit(-1);
    }

    inputfilename = (char*) malloc (256*sizeof(char));
    strcpy(inputfilename,argv[1]);

	printf("inputfilename=%s\n", inputfilename);
	fflush(0);

    if ((inputfile=fopen(inputfilename,"r")) == 0)
    {
		printf("Error openning input file.\n");
		exit(-1);
    }
    
    fscanf(inputfile, "%d\n", &dim); //Lê a dimensão das matrizes
    
    //Aloca as matrizes
    hipHostAlloc((void**)&A, dim * dim * sizeof(int), hipHostMallocMapped);
    hipHostAlloc((void**)&B, dim * dim * sizeof(int), hipHostMallocMapped); 
    hipHostAlloc((void**)&C, dim * dim * sizeof(int), hipHostMallocMapped); 
    hipHostAlloc((void**)&min_max_val, 3 * sizeof(int), hipHostMallocMapped); 

    //Aloca um vetor para armazenar os produtos escalares de cada linha
    // cudaHostAlloc((void*)&prod_escalar, sizeof(int), cudaHostAllocMapped);

     //Lê a matriz A
    for (i = 0; i < dim; i++)
        for (j = 0; j < dim; j++)
            fscanf(inputfile, "%d ",&(A[i*dim+j]));


    //Lê a matriz B
    for (i = 0; i < dim; i++)
        for(j = 0; j < dim; j++)
            fscanf(inputfile, "%d ",&(B[i*dim+j])); 

    min_max_val[0] = INT_MAX;
    min_max_val[1] = INT_MIN;

	hipDeviceSynchronize();
	
	// fecha o arquivo de entrada
	fclose(inputfile);





    int threadsPerBlock,blocksPerGrid; 

    //Define a quantidade de threads por bloco
    threadsPerBlock = 256; 
    
    //Define a quantidade de blocos por grade
    blocksPerGrid = (dim * dim + threadsPerBlock - 1) / threadsPerBlock; 




    //Invoca o kernel com blocksPerGrid blocos e threadsPerBlock threads
    mult <<<blocksPerGrid, threadsPerBlock>>> (A, B, C, dim * dim); 
    min_max <<<blocksPerGrid, threadsPerBlock>>> (A, B, min_max_val, dim * dim); 

	hipDeviceSynchronize();
	
    //Imprime o resultado no host
    int result = 0;
    for(i = 0; i < dim * dim; i++)
    {
        // printf("%d ",C[i]);
        result += C[i];
    }
    
    printf("%d\n", result);

    printf("%d %d", min_max_val[0], min_max_val[1]);
	
	

    //Desaloca os vetores no host e no device
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);
    hipHostFree(min_max_val);

}